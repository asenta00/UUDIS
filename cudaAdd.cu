
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
// broj niti koje �elimo koristiti
const int N = 10;
// kernel: funkcija koju izvr�avaju niti grafi�ke kartice
__global__ void add (int *a, int *b, int *c)
{
int tid = threadIdx.x;
c[tid] = a[tid] + b[tid];
}
int main()
{
// inicijaliziramo sat da mo�emo pratiti vrijeme izvr�avanja
 std::clock_t start;
 double duration;
 start = std::clock();
int a[N], b[N], c[N];
//pokaziva�i na memoriju na grafi�koj kartici
 int *dev_a, *dev_b, *dev_c;
// alociranje memorije na grafi�koj kartici
 hipMalloc( (void**)&dev_a, N * sizeof(int) );
 hipMalloc( (void**)&dev_b, N * sizeof(int) );
 hipMalloc( (void**)&dev_c, N * sizeof(int) );

// popunjavanje nizova a i b u glavnoj memoriji
 for (int i = 0; i < N; i++)
{
 a[i] = i;
b[i] = i;
 }
// kopiranje vrijednosti sa glavne memorije na memoriju grafi�ke kartice
 hipMemcpy ( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
 hipMemcpy ( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
// poziv kernela
// <<< broj blokova, broj niti po bloku>>>
// broj niti po bloku je maksimalno 512 zbog fizi�kih ograni�enja
 add<<<1,N>>>(dev_a, dev_b, dev_c);
 hipMemcpy ( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
 for (int i = 0; i < N; i++)
{
 std::cout << a[i] <<" "<< b[i] <<" "<< c[i] <<std::endl;
 }
 hipFree (dev_a);
hipFree (dev_b);
hipFree (dev_c);
// sat
 duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
 std::cout << "Time taken: " << duration << '\n';
 return 0;
}