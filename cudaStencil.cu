
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

const int N = 10;
using namespace std;

// N mora biti parametar jer GPU ne vidi konstante iz memorije
__global__ void stancil(float *in, float *out, int N)
{
  	// Nadopisati kod ovdje
	// Id niti
	int tid = threadIdx.x;
}

int main()
{
        std::clock_t start;
        double duration;
        start = std::clock();

        float *d_in, *d_out;
        float *h_in, *h_out;

        size_t size = N * sizeof(float);

        h_in = (float *) malloc(size);
        h_out = (float *) malloc(size);

        if ((h_in == NULL) || (h_out == NULL))
        {
                cout << "Greska prilikom stvaranja host vektora." << endl;
                return -1;
        }

        for (int i = 0; i < N; i++)
        {
                h_in[i] = i+1;
                h_out[i] = 0;
        }

        hipMalloc (&d_in, size);
        hipMalloc (&d_out, size);

        hipMemcpy (d_in, h_in, size, hipMemcpyHostToDevice);
        stancil<<<(N/512+1),512>>>(d_in, d_out, N);
        hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

        cout << "Rjesenje: " << endl;
        for (int i = 0; i < N; i++)
        {
                cout << h_in[i] << " " << h_out[i] << " " << endl;
        }

        hipFree (d_in);
        hipFree (d_out);

        duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
        cout << "Vrijeme izvrsavanja: "<< duration << endl;

        return 0;
}


